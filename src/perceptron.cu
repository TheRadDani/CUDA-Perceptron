#include "hip/hip_runtime.h"
#include <perceptron.h>

__global__ void perceptron(float *inputs, float *weights, float *output) {
    float sum = 0;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Calculate weighted sum
    for (int i = 0; i < NUM_INPUTS; i++) {
        sum += inputs[bid * NUM_INPUTS + i] * weights[tid * NUM_INPUTS + i];
    }
    sum += weights[tid * NUM_INPUTS + NUM_INPUTS];

    // Apply activation function
    if (sum > 0) {
        output[bid] = 1.0;
    } else {
        output[bid] = 0.0;
    }
}

void print_accuracy(float *output_h, float *expected_output_h, int num_examples) {
    float correct = 0;
    for (int i = 0; i < num_examples; i++) {
        if (output_h[i] == expected_output_h[i]) {
            correct++;
        }
    }
    float accuracy = correct / num_examples * 100;
    printf("Accuracy: %.2f%%\n", accuracy);
}


int main(int argc, char **argv) {
    float *inputs_h, *weights_h, *output_h;
    float *inputs_d, *weights_d, *output_d;
    int num_blocks, num_threads;

    // Allocate memory on host
    inputs_h = (float*)malloc(sizeof(float) * NUM_INPUTS * 4);
    weights_h = (float*)malloc(sizeof(float) * NUM_WEIGHTS * 2);
    output_h = (float*)malloc(sizeof(float) * 4);

    // Initialize inputs and weights
    inputs_h[0] = 0; inputs_h[1] = 0;
    inputs_h[2] = 0; inputs_h[3] = 1;
    inputs_h[4] = 1; inputs_h[5] = 0;
    inputs_h[6] = 1; inputs_h[7] = 1;

    weights_h[0] = 0.5; weights_h[1] = -0.5; weights_h[2] = 0.2;
    weights_h[3] = 0.9; weights_h[4] = 0.8; weights_h[5] = -0.1;

    // Allocate memory on device
    hipMalloc((void**)&inputs_d, sizeof(float) * NUM_INPUTS * 4);
    hipMalloc((void**)&weights_d, sizeof(float) * NUM_WEIGHTS * 2);
    hipMalloc((void**)&output_d, sizeof(float) * 4);

    // Copy inputs and weights to device
    hipMemcpy(inputs_d, inputs_h, sizeof(float) * NUM_INPUTS * 4, hipMemcpyHostToDevice);
    hipMemcpy(weights_d, weights_h, sizeof(float) * NUM_WEIGHTS * 2, hipMemcpyHostToDevice);

    // Set number of blocks and threads per block
    num_blocks = 2;
    num_threads = NUM_WEIGHTS;

    // Launch kernel
    perceptron<<<num_blocks, num_threads>>>(inputs_d, weights_d, output_d);

    // Copy output from device to host
    hipMemcpy(output_h, output_d, sizeof(float) * 4, hipMemcpyDeviceToHost);

    // Print output
    printf("Output: %f %f %f %f\n", output_h[0], output_h[1], output_h[2], output_h[3]);

    float expected_output_h[4] = {0, 0, 0, 1};
    print_accuracy(output_h, expected_output_h, 4);


    // Free memory on device
    hipFree(inputs_d);
    hipFree(weights_d);
    hipFree(output_d);

    // Free memory on host
    free(inputs_h);
    free(weights_h);
    free(output_h);

    return 0;
}
